#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib
// -lglut -o filter or (multicore lab) nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64
// -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may
// come but I call this version 1.0b2. 2017-12-04: Two fixes: Added
// command-lines (above), fixed a bug in computeImages that allocated too much
// memory. b3 2017-12-04: More fixes: Tightened up the kernel with edge
// clamping. Less code, nicer result (no borders). Cleaned up some messed up X
// and Y. b4 2022-12-07: A correction for a deprecated function.

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
#include <GLUT/glut.h>
#include <OpenGL/gl.h>
#else
#include <GL/glut.h>
#endif
#include "milli.h"
#include "readppm.h"

struct pixel {
  unsigned char r{}, g{}, b{};
};

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10
#define BLOCK_HEIGHT (maxKernelSizeX * 2 + 1)
#define BLOCK_WIDTH (maxKernelSizeY * 2 + 1)

__global__ void filter(pixel *image, pixel *out, const unsigned int imagesizex,
                       const unsigned int imagesizey, const int kernelsizex,
                       const int kernelsizey) {
  // map from blockIdx to pixel position
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ pixel block[BLOCK_HEIGHT][BLOCK_WIDTH];

  for (int dy{-kernelsizey}, i{}; dy <= kernelsizey; ++dy, ++i) {
    for (int dx{-kernelsizex}, j{}; dx <= kernelsizex; ++dx, ++j) {
      int yy{min(max(y + dy, 0), static_cast<int>(imagesizey) - 1)};
      int xx{min(max(x + dx, 0), static_cast<int>(imagesizex) - 1)};
      block[i][j] = image[yy * imagesizex + xx];
    }
  }

  int dy, dx;
  unsigned int sumx, sumy, sumz;

  int divby = (2 * kernelsizex + 1) *
              (2 * kernelsizey + 1);  // Works for box filters only!

  if (x < imagesizex && y < imagesizey)  // If inside image
  {
    // Filter kernel (simple box filter)
    sumx = 0;
    sumy = 0;
    sumz = 0;
    for (dy = -kernelsizey; dy <= kernelsizey; dy++)
      for (dx = -kernelsizex; dx <= kernelsizex; dx++) {
        // Use max and min to avoid branching!
        int yy = min(max(y + dy, 0), imagesizey - 1);
        int xx = min(max(x + dx, 0), imagesizex - 1);

        sumx += image[((yy)*imagesizex + (xx))].r;
        sumy += image[((yy)*imagesizex + (xx))].g;
        sumz += image[((yy)*imagesizex + (xx))].b;
      }
    out[(y * imagesizex + x)].r = sumx / divby;
    out[(y * imagesizex + x)].g = sumy / divby;
    out[(y * imagesizex + x)].b = sumz / divby;
  } else {
    printf("%d,%d is outside imagesize!", x, y);
  }
}

// Global variables for image data
pixel *dev_input, *dev_bitmap;
unsigned char *image, *pixels;
unsigned int imagesizey, imagesizex;  // Image size

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey) {
  if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY) {
    printf("Kernel size out of bounds!\n");
    return;
  }

  pixels = (unsigned char *)malloc(imagesizex * imagesizey * 3);
  hipMalloc((void **)&dev_input, imagesizex * imagesizey * 3);
  hipMemcpy(dev_input, image, imagesizey * imagesizex * 3,
             hipMemcpyHostToDevice);
  hipMalloc((void **)&dev_bitmap, imagesizex * imagesizey * 3);

  dim3 grid_dim{imagesizex, imagesizey};
  dim3 block_dim{1, 1};
  filter<<<grid_dim, block_dim>>>(dev_input, dev_bitmap, imagesizex, imagesizey,
                                  kernelsizex,
                                  kernelsizey);  // Awful load balance
  hipDeviceSynchronize();
  //	Check for errors!
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
  hipMemcpy(pixels, dev_bitmap, imagesizey * imagesizex * 3,
             hipMemcpyDeviceToHost);
  hipFree(dev_bitmap);
  hipFree(dev_input);
}

// Display images
void Draw() {
  // Dump the whole picture onto the screen.
  glClearColor(0.0, 0.0, 0.0, 1.0);
  glClear(GL_COLOR_BUFFER_BIT);

  if (imagesizey >= imagesizex) {  // Not wide - probably square. Original
                                   // left, result right.
    glRasterPos2f(-1, -1);
    glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image);
    glRasterPos2i(0, -1);
    glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels);
  } else {  // Wide image! Original on top, result below.
    glRasterPos2f(-1, -1);
    glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    glRasterPos2i(-1, 0);
    glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image);
  }
  glFlush();
}

// Main program, inits
int main(int argc, char **argv) {
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);

  if (argc > 1)
    image = readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey);
  else
    image = readppm((char *)"maskros512.ppm", (int *)&imagesizex,
                    (int *)&imagesizey);

  if (imagesizey >= imagesizex)
    glutInitWindowSize(imagesizex * 2, imagesizey);
  else
    glutInitWindowSize(imagesizex, imagesizey * 2);
  glutCreateWindow("Lab 5");
  glutDisplayFunc(Draw);

  ResetMilli();

  computeImages(2, 2);

  // You can save the result to a file like this:
  //	writeppm("out.ppm", imagesizey, imagesizex, pixels);

  glutMainLoop();
  return 0;
}
