#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib
// -lglut -o filter or (multicore lab) nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64
// -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may
// come but I call this version 1.0b2. 2017-12-04: Two fixes: Added
// command-lines (above), fixed a bug in computeImages that allocated too much
// memory. b3 2017-12-04: More fixes: Tightened up the kernel with edge
// clamping. Less code, nicer result (no borders). Cleaned up some messed up X
// and Y. b4 2022-12-07: A correction for a deprecated function.

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
#include <GLUT/glut.h>
#include <OpenGL/gl.h>
#else
#include <GL/glut.h>
#endif
#include "milli.h"
#include "readppm.h"

struct pixel {
  unsigned char r{}, g{}, b{};
};

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10

#define BLOCKX 2
#define BLOCKY 2

#define BLOCK_HEIGHT (maxKernelSizeX * 2 + 1)
#define BLOCK_WIDTH (maxKernelSizeY * 2 + 1)

__global__ void box(pixel *image, pixel *out, const unsigned int imagesizex,
                    const unsigned int imagesizey, const int kernelsizex,
                    const int kernelsizey) {
  __shared__ pixel
      block[BLOCKY + 2 * maxKernelSizeY][BLOCKX + 2 * maxKernelSizeX];

  // How many pixels we will need for the filter in our block
  int width{BLOCKX + 2 * kernelsizex};
  int height{BLOCKY + 2 * kernelsizey};

  int chunksizey{height / BLOCKY};
  int starty{static_cast<int>(threadIdx.y) * chunksizey};
  int endy{starty + chunksizey};
  if (threadIdx.y == BLOCKY - 1) endy += height % BLOCKY;

  int chunksizex{width / BLOCKX};
  int startx{static_cast<int>(threadIdx.x) * chunksizex};
  int endx{startx + chunksizex};
  if (threadIdx.x == BLOCKX - 1) endx += width % BLOCKX;

  // The top left corner of our block in the image
  int firsty = blockIdx.y * blockDim.y;
  int firstx = blockIdx.x * blockDim.x;

  for (int i{starty}; i <= endy; ++i) {
    for (int j{startx}; j <= endx; ++j) {
      int imgy{-kernelsizey + i + firsty};
      int imgx{-kernelsizex + j + firstx};
      int yy{min(max(imgy, 0), static_cast<int>(imagesizey) - 1)};
      int xx{min(max(imgx, 0), static_cast<int>(imagesizex) - 1)};
      block[i][j] = image[yy * imagesizex + xx];
    }
  }

  // Synch all threads in block
  __syncthreads();

  int kernel_height{2 * kernelsizey + 1};
  int kernel_width{2 * kernelsizex + 1};

  // Filter kernel (simple box filter)
  unsigned sumx{}, sumy{}, sumz{};
  for (size_t i{}; i < kernel_height; ++i) {
    for (size_t j{}; j < kernel_width; ++j) {
      pixel p{block[i + threadIdx.y][j + threadIdx.x]};
      sumx += p.r;
      sumy += p.g;
      sumz += p.b;
    }
  }

  // map from blockIdx to pixel position
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // Works for box filters only!
  int divby{(2 * kernelsizex + 1) * (2 * kernelsizey + 1)};
  out[y * imagesizex + x].r = sumx / divby;
  out[y * imagesizex + x].g = sumy / divby;
  out[y * imagesizex + x].b = sumz / divby;
}

__global__ void gaussian(pixel *image, pixel *out,
                         const unsigned int imagesizex,
                         const unsigned int imagesizey, const int kernelsizex,
                         const int kernelsizey) {
  __shared__ pixel
      block[BLOCKY + 2 * maxKernelSizeY][BLOCKX + 2 * maxKernelSizeX];

  // How many pixels we will need for the filter in our block
  int width{BLOCKX + 2 * kernelsizex};
  int height{BLOCKY + 2 * kernelsizey};

  int chunksizey{height / BLOCKY};
  int starty{static_cast<int>(threadIdx.y) * chunksizey};
  int endy{starty + chunksizey};
  if (threadIdx.y == BLOCKY - 1) endy += height % BLOCKY;

  int chunksizex{width / BLOCKX};
  int startx{static_cast<int>(threadIdx.x) * chunksizex};
  int endx{startx + chunksizex};
  if (threadIdx.x == BLOCKX - 1) endx += width % BLOCKX;

  // The top left corner of our block in the image
  int firsty = blockIdx.y * blockDim.y;
  int firstx = blockIdx.x * blockDim.x;

  for (int i{starty}; i <= endy; ++i) {
    for (int j{startx}; j <= endx; ++j) {
      int imgy{-kernelsizey + i + firsty};
      int imgx{-kernelsizex + j + firstx};
      int yy{min(max(imgy, 0), static_cast<int>(imagesizey) - 1)};
      int xx{min(max(imgx, 0), static_cast<int>(imagesizex) - 1)};
      block[i][j] = image[yy * imagesizex + xx];
    }
  }

  // Synch all threads in block
  __syncthreads();

  int kernel_height{2 * kernelsizey + 1};
  int kernel_width{2 * kernelsizex + 1};

  int gaussian[5][5]{{1, 4, 6, 4, 1},
                     {4, 16, 24, 16, 4},
                     {6, 24, 36, 24, 6},
                     {4, 16, 24, 16, 4},
                     {1, 4, 6, 4, 1}};

  int divby{};
  unsigned sumx{}, sumy{}, sumz{};
  for (size_t i{}; i < kernel_height; ++i) {
    for (size_t j{}; j < kernel_width; ++j) {
      pixel p{block[i + threadIdx.y][j + threadIdx.x]};
      sumx += p.r * gaussian[i][j];
      sumy += p.g * gaussian[i][j];
      sumz += p.b * gaussian[i][j];
      divby += gaussian[i][j];
    }
  }

  // map from blockIdx to pixel position
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  out[y * imagesizex + x].r = static_cast<float>(sumx) / divby;
  out[y * imagesizex + x].g = static_cast<float>(sumy) / divby;
  out[y * imagesizex + x].b = static_cast<float>(sumz) / divby;
}

// Global variables for image data
pixel *dev_input, *dev_bitmap;
pixel *image, *pixels;
// unsigned char *image, *pixels;
unsigned int imagesizey, imagesizex;  // Image size

enum filtertype {
  Gaussian,
  Box,
  Median,
};

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey, bool seperate,
                   filtertype ft) {
  if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY) {
    printf("Kernel size out of bounds!\n");
    return;
  }

  hipMalloc(&dev_input, imagesizex * imagesizey * 3);
  hipMemcpy(dev_input, image, imagesizey * imagesizex * 3,
             hipMemcpyHostToDevice);
  hipMalloc(&dev_bitmap, imagesizex * imagesizey * 3);

  dim3 block_dim{BLOCKX, BLOCKY};
  dim3 grid_dim{imagesizex / block_dim.x, imagesizey / block_dim.y};

  if (ft == Box) {
    box<<<grid_dim, block_dim>>>(dev_input, dev_bitmap, imagesizex, imagesizey,
                                 kernelsizex, seperate ? 0 : kernelsizey);
  } else if (ft == Gaussian) {
    gaussian<<<grid_dim, block_dim>>>(dev_input, dev_bitmap, imagesizex,
                                      imagesizey, kernelsizex,
                                      seperate ? 0 : kernelsizey);
  }
  hipDeviceSynchronize();

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));

  pixels = new pixel[imagesizex * imagesizey];
  hipMemcpy(pixels, dev_bitmap, imagesizey * imagesizex * 3,
             hipMemcpyDeviceToHost);
  if (seperate) {
    hipMemcpy(dev_input, pixels, imagesizey * imagesizex * 3,
               hipMemcpyHostToDevice);

    if (ft == Box) {
      box<<<grid_dim, block_dim>>>(dev_input, dev_bitmap, imagesizex,
                                   imagesizey, 0, kernelsizey);
    } else if (ft == Gaussian) {
      gaussian<<<grid_dim, block_dim>>>(dev_input, dev_bitmap, imagesizex,
                                        imagesizey, kernelsizex,
                                        seperate ? 0 : kernelsizey);
    }
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy(pixels, dev_bitmap, imagesizey * imagesizex * 3,
               hipMemcpyDeviceToHost);
  }

  hipFree(dev_bitmap);
  hipFree(dev_input);
}

// Display images
void Draw() {
  // Dump the whole picture onto the screen.
  glClearColor(0.0, 0.0, 0.0, 1.0);
  glClear(GL_COLOR_BUFFER_BIT);

  if (imagesizey >= imagesizex) {  // Not wide - probably square. Original
                                   // left, result right.
    glRasterPos2f(-1, -1);
    glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image);
    glRasterPos2i(0, -1);
    glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels);
  } else {  // Wide image! Original on top, result below.
    glRasterPos2f(-1, -1);
    glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    glRasterPos2i(-1, 0);
    glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image);
  }
  glFlush();
}

// Main program, inits
int main(int argc, char **argv) {
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);

  if (argc > 1)
    image = reinterpret_cast<pixel *>(
        readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey));
  else
    image = reinterpret_cast<pixel *>(readppm(
        (char *)"maskros512.ppm", (int *)&imagesizex, (int *)&imagesizey));

  if (imagesizey >= imagesizex)
    glutInitWindowSize(imagesizex * 2, imagesizey);
  else
    glutInitWindowSize(imagesizex, imagesizey * 2);
  glutCreateWindow("Lab 5");
  glutDisplayFunc(Draw);

  ResetMilli();

  // computeImages(10, 10, false);
  computeImages(2, 2, true, Gaussian);

  // You can save the result to a file like this:
  //	writeppm("out.ppm", imagesizey, imagesizex, pixels);

  glutMainLoop();
  return 0;
}
